/* Name: Georgios Krommydas
 * A.M.: 3260
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/* 
 * Retrieves and prints information for every installed NVIDIA
 * GPU device
 */
void cuinfo_print_devinfo()
{
	int num_devs, i;
	hipDeviceProp_t dev_prop;
	
	hipGetDeviceCount(&num_devs);
	if (num_devs == 0)
	{
		printf("No CUDA devices found.\n");
		return;
	}

	for (i = 0; i < num_devs; i++)
	{
		/* TODO: Retrieve and pretty-print all the necessary information */
		hipGetDeviceProperties(&dev_prop, i);
		printf("-------------------------------------------------------------\n");	
		printf("Information for CUDA Device - %d\n", i+1);
		printf("-------------------------------------------------------------\n");	
		printf("Device Name: %s\n", dev_prop.name);
		printf("Device CUDA compute capability: %d.%d\n", dev_prop.major, dev_prop.minor);
		printf("Device number of streaming multiprocessors: %d\n", dev_prop.multiProcessorCount);
		printf("Device max number of threads per block: %d\n", dev_prop.maxThreadsPerBlock);
		printf("Device size of global memory: %lu bytes (= 24 Gbytes)\n", dev_prop.totalGlobalMem);
		printf("Device size of shared memory per block: %lu bytes\n", dev_prop.sharedMemPerBlock);
		printf("-------------------------------------------------------------\n");
	}
}

int main()
{
	cuinfo_print_devinfo();
	return 0;
}
